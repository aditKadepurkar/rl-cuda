#include "hip/hip_runtime.h"
/**
 * ppo.cu
 * 
 * 
 */


#include "ppo.h"

PPO::PPO(Environment* env) {
    // setup a default network architecture
    std::vector<int> layers = {128, 256, 128, 64};

    this->policy_network = new MLP(layers);
    this->value_network = new MLP(layers);
    this->env = env;
    this->env_max_steps = env->max_steps;
    this->rollout_buffer = std::vector<Rollout>(ROLLOUT_BUFFER_SIZE);

}

PPO::PPO(Environment* env, std::vector<int> policy_network_dims, std::vector<int> value_network_dims) {

    this->policy_network = new MLP(policy_network_dims);
    this->value_network = new MLP(value_network_dims);
    this->env = env;
    this->env_max_steps = env->max_steps;
    this->rollout_buffer = std::vector<Rollout>(ROLLOUT_BUFFER_SIZE);
}


void PPO::train(int num_timesteps) {

    std::cout << "Training PPO for " << num_timesteps << " timesteps" << std::endl;

    // collect rollouts -- will put this in a loop later, testing for now
    collect_rollouts(); // collect_rollouts


    std::cout << "Collected rollouts " << rollout_buffer.size() << std::endl;



    /* The followign code shows how to access the data in the rollout buffer
    
    
    

    int state_size = env->state_size;

    std::vector<float> example_state(state_size);
    hipMemcpy(example_state.data(), rollout_buffer[0].states, state_size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "Example state: ";
    for (const auto& val : example_state) {
        std::cout << val << " ";
    }
    std::cout << std::endl;
    */


    // now we do some training!!!!

    /* Basically my notes sheet for the next steps:
    - I currently have the rollouts which have all the following:
        - states
        - actions
        - rewards
        - values
        - log_probs
        - advantages
        - returns

    Now I just need to do the calculations? I think?
    
    
    */

    float *d_states;
    float *d_output;
    float *d_new_log_probs;
    float *d_old_log_probs;

    Rollout& rollout = rollout_buffer[0];


    hipMalloc(&d_old_log_probs, env_max_steps * sizeof(float));
    hipMalloc(&d_states, env_max_steps * env->state_size * sizeof(float));
    hipMalloc(&d_output, policy_network->output_dim * sizeof(float));
    policy_network->forward(rollout.states, d_output);
    policy_network->log_prob(d_output, rollout.actions, d_old_log_probs);

    hipMalloc(&d_new_log_probs, sizeof(float) * policy_network->output_dim);

    // int state_size = env->state_size;
    // int action_dim = env->action_size;

    for (int i = 0; i < ROLLOUT_BUFFER_SIZE; ++i) {
        std::cout << "Training on rollout " << i << std::endl;

        rollout = rollout_buffer[i];

        // get new log probs
        policy_network->forward(rollout.states, d_output);
        policy_network->log_prob(d_output, rollout.actions, d_new_log_probs);

        // r_t(theta) = pi_theta(a_t | s_t) / pi_theta_old(a_t | s_t)
        float *d_ratios;
        hipMalloc(&d_ratios, env_max_steps * sizeof(float));
        policy_network->ratio(d_new_log_probs, d_old_log_probs, d_ratios);


        float *d_surrogate;
        hipMalloc(&d_surrogate, env_max_steps * sizeof(float));
        policy_network->surrogate_loss(d_ratios, rollout.advantages, d_surrogate);


        // update the parameters 
        policy_network->backward(d_surrogate, policy_network->learning_rate);

        float *d_value_grads;
        hipMalloc(&d_value_grads, env_max_steps * sizeof(float));

        // caclulating the value grad with teh new func i made which I hope works correctly
        compute_value_grad(rollout.values, rollout.returns, d_value_grads, env_max_steps);

        value_network->backward(d_value_grads, value_network->learning_rate);


        // print d_value_grads and d_surrogate
        std::vector<float> h_value_grads(env_max_steps);
        std::vector<float> h_surrogate(env_max_steps);

        hipMemcpy(h_value_grads.data(), d_value_grads, env_max_steps * sizeof(float), hipMemcpyDeviceToHost);
        hipMemcpy(h_surrogate.data(), d_surrogate, env_max_steps * sizeof(float), hipMemcpyDeviceToHost);

        // std::cout << "Value grads: ";
        // for (const auto& val : h_value_grads) {
        //     std::cout << val << " ";
        // }
        // std::cout << std::endl;

        std::cout << "Surrogate: ";
        for (const auto& val : h_surrogate) {
            std::cout << val << " ";
        }
        std::cout << std::endl;

        // Free allocated memory
        hipFree(d_ratios);
        hipFree(d_surrogate);
    }




}

// #define CUDA_CHECK(call) \
//     do { \
//         hipError_t err = call; \
//         if (err != hipSuccess) { \
//             fprintf(stderr, "CUDA error in file '%s' in line %i : %s.\n", \
//                     __FILE__, __LINE__, hipGetErrorString(err)); \
//             exit(EXIT_FAILURE); \
//         } \
//     } while (0)

void PPO::collect_rollouts() {
    int action_dim = env->action_size;
    int state_size = env->state_size;

    for (int i = 0; i < ROLLOUT_BUFFER_SIZE; ++i) {
        Rollout& rollout = rollout_buffer[i];

        if (rollout.states == nullptr) {
            hipMalloc(&rollout.states, env_max_steps * state_size * sizeof(float));
            hipMalloc(&rollout.actions, env_max_steps * action_dim * sizeof(float));
            hipMalloc(&rollout.rewards, env_max_steps * sizeof(float));
            hipMalloc(&rollout.values, env_max_steps * sizeof(float));
            // CUDA_CHECK(hipMalloc(&rollout.log_probs, env_max_steps * sizeof(float)));
            hipMalloc(&rollout.advantages, env_max_steps * sizeof(float));
            hipMalloc(&rollout.returns, env_max_steps * sizeof(float));
        }

        float* d_state;
        float* d_next_state;
        float* d_reward;
        bool* d_done;
        float* d_action;
        float* d_value;

        bool h_done;

        hipMalloc(&d_state, state_size * sizeof(float));
        hipMalloc(&d_next_state, state_size * sizeof(float));
        hipMalloc(&d_reward, sizeof(float));
        hipMalloc(&d_done, sizeof(bool));
        hipMalloc(&d_action, action_dim * sizeof(float));
        hipMalloc(&d_value, sizeof(float));

        int step = 0;
        h_done = false;
        env->reset(d_state);

        // std::cout << "Rollout " << i << std::endl;

        while (!h_done && step < env_max_steps) {
            // Generate action directly on device for now
            hipMemset(d_action, 0.1, action_dim * sizeof(float));

            env->step(d_action, d_next_state, d_reward, d_done);

            value_network->forward(d_state, d_value);

            // Store rollout data on the device
            hipMemcpy(&rollout.states[step * state_size], d_state, state_size * sizeof(float), hipMemcpyDeviceToDevice);
            hipMemcpy(&rollout.actions[step * action_dim], d_action, action_dim * sizeof(float), hipMemcpyDeviceToDevice);
            hipMemcpy(&rollout.rewards[step], d_reward, sizeof(float), hipMemcpyDeviceToDevice);
            hipMemcpy(&rollout.values[step], d_value, sizeof(float), hipMemcpyDeviceToDevice);

            hipMemcpy(&h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost);

            hipMemcpy(d_state, d_next_state, state_size * sizeof(float), hipMemcpyDeviceToDevice);
            step++;
        }

        // std::cout << "Step " << step << std::endl;

        // Compute advantages, values, etc. on the device
        for (int k = 0; k < step; k++) {
            float reward, value, next_value;

            hipMemcpy(&reward, &rollout.rewards[k], sizeof(float), hipMemcpyDeviceToHost);
            hipMemcpy(&value, &rollout.values[k], sizeof(float), hipMemcpyDeviceToHost);
            // std::cout << "Value: " << value << std::endl;

            // next_value = (k == step - 1) ? 0 : rollout.values[k + 1];
            
            if (k == step - 1) {
                next_value = 0;
            } else {
                hipMemcpy(&next_value, &rollout.values[k + 1], sizeof(float), hipMemcpyDeviceToHost);
            }


            // these are hyperparameters that I should probably make into variables
            // but for now they are hardcoded
            float delta = reward + 0.99 * next_value - value;
            


            // lotss of optimization that can be done here very easily by not wasting all the time copying back and forth
            // TODO later

            float advantage;
            if (k == 0) {
                advantage = delta;
            } else {
                float prev_advantage;
                hipMemcpy(&prev_advantage, &rollout.advantages[k - 1], sizeof(float), hipMemcpyDeviceToHost);
                advantage = delta + 0.99 * 0.95 * prev_advantage;
            }
            hipMemcpy(&rollout.advantages[k], &advantage, sizeof(float), hipMemcpyHostToDevice);

            float ret;
            if (k == 0) {
                ret = reward;
            } else {
                float prev_return;
                hipMemcpy(&prev_return, &rollout.returns[k - 1], sizeof(float), hipMemcpyDeviceToHost);
                ret = reward + 0.99 * prev_return;
            }
            hipMemcpy(&rollout.returns[k], &ret, sizeof(float), hipMemcpyHostToDevice);
        }


        step = 0;

        // Free device memory for this rollout
        hipFree(d_state);
        hipFree(d_next_state);
        hipFree(d_reward);
        hipFree(d_done);
        hipFree(d_action);
    }
}