/**
 * network.cu
 * 
 * 
 */


#include "network.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <hiprand.h>



MLP::MLP(const std::vector<int>& layers) : layerSizes(layers) {
    hipdnnCreate(&cudnn);
    hipblasCreate(&cublas);
    numLayers = layers.size() - 1;

    initializeLayers();
}

void MLP::initializeLayers() {
    hiprandGenerator_t randGen;
    hiprandCreateGenerator(&randGen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(randGen, 1234ULL);

    for (int i = 0; i < numLayers; ++i) {
        int inputSize = layerSizes[i];
        int outputSize = layerSizes[i + 1];

        float *d_weights, *d_biases, *d_activations;
        hipMalloc(&d_weights, inputSize * outputSize * sizeof(float));
        hipMalloc(&d_biases, outputSize * sizeof(float));
        hipMalloc(&d_activations, outputSize * sizeof(float));

        // Initialize weights and biases with random values
        hiprandGenerateUniform(randGen, d_weights, inputSize * outputSize);
        hiprandGenerateUniform(randGen, d_biases, outputSize);

        weights.push_back(d_weights);
        biases.push_back(d_biases);
        activations.push_back(d_activations);
    }

    hiprandDestroyGenerator(randGen);
}

void MLP::forward(float* input, float* output) {

    // printing all the values in activations
    for (int i : layerSizes) {
        std::cout << i << std::endl;
    }

    float* currInput = input;
    float alpha = 1.0f, beta = 0.0f;

    for (int i = 0; i < numLayers; ++i) {
        int inputSize = layerSizes[i];
        int outputSize = layerSizes[i + 1];

        // Perform matrix multiplication: Y = W * X + B
        hipblasSgemm(
            cublas, HIPBLAS_OP_N, HIPBLAS_OP_N,
            outputSize, 1, inputSize,
            &alpha,
            weights[i], outputSize, // W: (outputSize, inputSize)
            currInput, inputSize,   // X: (inputSize, 1)
            &beta,
            activations[i], outputSize // Y: (outputSize, 1)
        );

        hipblasSaxpy(cublas, outputSize, &alpha, biases[i], 1, activations[i], 1);

        currInput = activations[i];
    }

    hipMemcpy(output, activations.back(), layerSizes.back() * sizeof(float), hipMemcpyDeviceToHost);
}

MLP::~MLP() {
    cleanup();
    hipdnnDestroy(cudnn);
    hipblasDestroy(cublas);
}

void MLP::cleanup() {
    for (auto& w : weights) hipFree(w);
    for (auto& b : biases) hipFree(b);
    for (auto& a : activations) hipFree(a);
}
