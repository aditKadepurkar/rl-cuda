#include "env/env.h"
#include <iostream>
#include <cstdlib>
#include <iomanip>

#define NUM_STEPS 100

int main() {
    int state_size = 10;
    int action_dim = 5;

    Environment env(state_size, action_dim);

    float* d_state;
    float* d_next_state;
    float* d_reward;
    bool* d_done;
    float* d_action;

    hipMalloc(&d_state, state_size * sizeof(float));
    hipMalloc(&d_next_state, state_size * sizeof(float));
    hipMalloc(&d_reward, sizeof(float));
    hipMalloc(&d_done, sizeof(bool));
    hipMalloc(&d_action, action_dim * sizeof(float));



    env.reset(d_state);

    std::cout << "Environment reset complete." << std::endl;

    float h_state[state_size];
    float h_action[action_dim];

    for (int i = 0; i < NUM_STEPS; i++) {

        for (int j = 0; j < action_dim; j++) {
            h_action[j] = static_cast<float>(rand()) / RAND_MAX;
        }
        hipMemcpy(d_action, h_action, action_dim * sizeof(float), hipMemcpyHostToDevice);


        env.step(d_action, d_next_state, d_reward, d_done);

        hipMemcpy(h_state, d_state, state_size * sizeof(float), hipMemcpyDeviceToHost);

        std::cout << "Step " << i + 1 << " | Action: [ ";
        for (int j = 0; j < action_dim; j++) {
            std::cout << std::fixed << std::setprecision(4) << h_action[j] << " ";
        }
        std::cout << "] | State: [ ";
        for (int j = 0; j < state_size; j++) {
            std::cout << std::fixed << std::setprecision(4) << h_state[j] << " ";
        }
        std::cout << "]" << std::endl;

        bool h_done;
        hipMemcpy(&h_done, d_done, sizeof(bool), hipMemcpyDeviceToHost);

        if (h_done) {
            std::cout << "Episode finished early at step " << i + 1 << std::endl;
            break;
        }

        hipMemcpy(d_state, d_next_state, state_size * sizeof(float), hipMemcpyDeviceToDevice);
    }

    hipFree(d_state);
    hipFree(d_next_state);
    hipFree(d_reward);
    hipFree(d_done);

    std::cout << "Simulation complete." << std::endl;
    return 0;
}
